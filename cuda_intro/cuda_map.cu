
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>


__global__ void _cuda_parallel_multiplication(int count, int* test_data, int magnitude);

int main() {
    int count = 60000000;    // 60 million elements
    int* test_data = new int[count];

    for(int i = 0; i < count; i++)
        test_data[i] = i;

    // Perform calculation on host CPU
    auto t1 = std::chrono::high_resolution_clock::now();
    for(int i = 0; i < count; i++)
        test_data[i] = test_data[i] * 5;
    auto t2 = std::chrono::high_resolution_clock::now();
    
    // Copy data to device
    int* d_test_data;
    hipMalloc(&d_test_data, count * sizeof(int));
    hipMemcpy(d_test_data, test_data, count * sizeof(int), hipMemcpyHostToDevice);
    
    // Launch kernel
    int block_count = ceil((double)count / 1024);
    _cuda_parallel_multiplication<<<block_count, 1024>>>(count, d_test_data, 5);
    
    hipDeviceSynchronize();
    hipMemcpy(test_data, d_test_data, count * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_test_data);

    for(int i = 0; i < 10; i++)
        std::cout << i << ": " << test_data[i] << std::endl;

    // Copy results back to device
    std::cout	<< "CPU time: "
                << std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count()
                << "ms" << std::endl;
}


__global__ void _cuda_parallel_multiplication(int count, int* test_data, int magnitude) {

     int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

     if (globalIdx < count)
        test_data[globalIdx] = test_data[globalIdx] * magnitude;

}


